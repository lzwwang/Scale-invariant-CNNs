#include "hip/hip_runtime.h"
// Implemented by Angjoo Kanazawa, Abhishek Sharma 2013
#include <cstdlib>
#include <cstring>
#include <stdio.h>
#include <thrust/device_vector.h>
#include <thrust/count.h>

#include "caffe/common.hpp"
#include "caffe/util/transformation.hpp"
#include "caffe/blob.hpp"

namespace caffe {

template <typename Dtype>
__global__ void
nn_interpolation_kernel(const int nthreads, const Dtype *oldDPtr,
                        const int oldSheetCount, Dtype *newDPtr,
                        const int newSheetCount, const float *coord) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < nthreads) {
    int offset = index % newSheetCount;
    int numSheet = index / newSheetCount;
    int backSheetOffset = static_cast<int>(coord[offset]);
    if (backSheetOffset >= 0) {
      newDPtr[numSheet * newSheetCount + offset] =
          oldDPtr[numSheet * oldSheetCount + backSheetOffset];
    } else {
      newDPtr[numSheet * newSheetCount + offset] = 0;
    }
  }
}

template <typename Dtype>
__global__ void bilinear_interpolation_kernel(
    const int nthreads, const Dtype *oldDPtr, const int oldSheetCount,
    Dtype *newDPtr, const int newSheetCount, const float *coord, const int W) {
  // need W
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < nthreads) {
    int offset = index % newSheetCount; // p00: r0*W + c0
    int numSheet = index / newSheetCount;
    int backSheetOffset = static_cast<int>(coord[offset]);
    if (backSheetOffset >= 0) {
      int c0 = backSheetOffset % W;
      // p11: r1*W + c1
      int ind_p11 = static_cast<int>(coord[offset + newSheetCount]);
      int c1 = ind_p11 % W;

      int ind_p01 = backSheetOffset - c0 + c1; // r0*W + c1
      int ind_p10 = ind_p11 - c1 + c0;         // r1*W  + c0

      float dc = coord[offset + 2 * newSheetCount];
      float dr = coord[offset + 3 * newSheetCount];

      float w00 = (1 - dc) * (1 - dr);
      float w01 = (1 - dr) * dc;
      float w10 = (1 - dc) * dr;
      float w11 = dr * dc;

      int bigOffset = numSheet * oldSheetCount;
      newDPtr[numSheet * newSheetCount + offset] =
          w00 * oldDPtr[bigOffset + backSheetOffset] +
          w01 * oldDPtr[bigOffset + ind_p01] +
          w10 * oldDPtr[bigOffset + ind_p10] +
          w11 * oldDPtr[bigOffset + ind_p11];
    } else {
      newDPtr[numSheet * newSheetCount + offset] = 0;
    }
  }
}

template <typename Dtype>
void InterpImageNN_gpu(const Blob<Dtype> *orig, const float *coord,
                       Blob<Dtype> *warped, const Interp &interp) {
  // Get the parameters from the original and warped and apply the
  // transformation to it.
  const Dtype *orgDataPtr = orig->gpu_data();
  Dtype *warpedDataPtr = warped->mutable_gpu_data();
  int oldNPerSheet = orig->height() * orig->width();
  int newNPerSheet = warped->height() * warped->width();
  int nCount = warped->count();
  switch (interp) {
  case NN:
    nn_interpolation_kernel<Dtype> << <CAFFE_GET_BLOCKS(nCount),
                                       CAFFE_CUDA_NUM_THREADS>>>
        (nCount, orgDataPtr, oldNPerSheet, warpedDataPtr, newNPerSheet, coord);
    break;
  case BILINEAR:
    bilinear_interpolation_kernel<Dtype> << <CAFFE_GET_BLOCKS(nCount),
                                             CAFFE_CUDA_NUM_THREADS>>>
        (nCount, orgDataPtr, oldNPerSheet, warpedDataPtr, newNPerSheet, coord,
         orig->width());
    break;
  default:
    LOG(ERROR) << "Unknown interpolation mode " << interp;
  }
  CUDA_POST_KERNEL_CHECK;
}

// Explicit instantiation
template void InterpImageNN_gpu<float>(const Blob<float> *orig,
                                       const float *coord, Blob<float> *warped,
                                       const Interp &interp);
// template void InterpImageNN_gpu<double>(const Blob<double> *orig,
//                                         const float *coord,
//                                         Blob<double> *warped,
//                                         const Interp &intep);

/********************** PropagateErrorNN_gpu ***************************/
// If we kernalize eahc pixel in the top (warped image), bc of race conditions
// we need to use atomicAdd, but it's slow and there is no double implementation
// of atomicAdd.
// So instead, parallelize over each pixel in the bottom (original) and for each
// pixel loop over te coord, find those top neurons that came from this bottom
// pixel and add. Similar to MaxPoolBackward
// Super. fucking. slow. duh.
template <typename Dtype>
__global__ void PropagateErrorNN_kernel_nonatomic(
    const int nthreads, const Dtype *top_diff, const int width,
    const int height, const int channels, const int num, const int top_len,
    const float *coord, Dtype *bottom_diff) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < nthreads) {
    // find out the target index to look for in coord
    // can do this the way abhishek did so we can save on some computation (like
    // with SheetCount)
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;

    int target_ind = h * width + w;
    // move over top_diff ptr to the beginning of its h x w sheet:
    // top_len = width_top * height_top
    top_diff += (n * channels + c) * top_len;

    Dtype gradient = 0;
    // loop over coord and add to grad IF coord[i] == target_ind
    for (int i = 0; i < top_len; ++i) {
      gradient += top_diff[i] * (static_cast<int>(coord[i]) == target_ind);
    }

    bottom_diff[index] += gradient;
  }
}

template <typename Dtype>
__global__ void nn_propagation_kernel(int nThreads, const Dtype *topDataPtr,
                                      int topSheetCount, Dtype *bottomDataPtr,
                                      int bottomSheetCount,
                                      const float *coord) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < nThreads) {
    int offset = index % topSheetCount;
    int numSheet = index / topSheetCount;

    int bottomSheetOffset = static_cast<int>(coord[offset]);
    if (bottomSheetOffset >= 0) {
      int bottomFinalOffset = numSheet * bottomSheetCount + bottomSheetOffset;
      // AJ: as atomicAdd is only available to float, this only works if
      // Dtype=float
      atomicAdd(
          (&bottomDataPtr[bottomFinalOffset]),
          static_cast<float>(topDataPtr[numSheet * topSheetCount + offset]));
    }
  }
}

template <typename Dtype>
__global__ void
bilinear_propagation_kernel(int nThreads, const Dtype *topDataPtr,
                            int topSheetCount, Dtype *bottomDataPtr,
                            int bottomSheetCount, const float *coord, int W) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < nThreads) {
    int offset = index % topSheetCount;
    int numSheet = index / topSheetCount;
    int bottomSheetOffset = static_cast<int>(coord[offset]);
    if (bottomSheetOffset >= 0) {
      int c0 = bottomSheetOffset % W;
      int ind_p11 = static_cast<int>(coord[offset + topSheetCount]);
      int c1 = ind_p11 % W;

      int ind_p01 = bottomSheetOffset - c0 + c1; // r0*W + c1
      int ind_p10 = ind_p11 - c1 + c0;           // r1*W  + c0

      float dc = coord[offset + 2 * topSheetCount];
      float dr = coord[offset + 3 * topSheetCount];

      // printf("@%d offset %d==>p00 %d c0 %d c1 %d ind_p11 %d dc %.2f dr
      // %.2f\n", index, offset,
      //        bottomSheetOffset, c0, c1, ind_p11, dc, dr);

      float w00 = (1 - dc) * (1 - dr);
      float w01 = (1 - dr) * dc;
      float w10 = (1 - dc) * dr;
      float w11 = dr * dc;

      // float top_error = static_cast<float>(topDataPtr[numSheet*topSheetCount
      // + offset]);
      float top_error = static_cast<float>(topDataPtr[index]);

      int commonOffset = numSheet * bottomSheetCount;

      // p00
      atomicAdd((&bottomDataPtr[commonOffset + bottomSheetOffset]),
                w00 * top_error);
      // p01
      atomicAdd((&bottomDataPtr[commonOffset + ind_p01]), w01 * top_error);
      // p10
      atomicAdd((&bottomDataPtr[commonOffset + ind_p10]), w10 * top_error);
      // p11
      atomicAdd((&bottomDataPtr[commonOffset + ind_p11]), w11 * top_error);
    }
  }
}

template <typename Dtype>
void PropagateErrorNN_gpu(const Blob<Dtype> *top, const float *coord,
                          Blob<Dtype> *bottom, const Interp &interp) {
  // Get the parameters from the original and warped and apply the
  // transformation to it.
  const Dtype *topDataPtr = top->gpu_diff();
  Dtype *bottomDataPtr = bottom->mutable_gpu_diff();
  int topNPerSheet = top->height() * top->width();
  int bottomNPerSheet = bottom->height() * bottom->width();
  // atomicAdd needs nTop many threads
  int nCount = top->count();
  switch (interp) {
  case NN:
    nn_propagation_kernel<Dtype> << <CAFFE_GET_BLOCKS(nCount),
                                     CAFFE_CUDA_NUM_THREADS>>>
        (nCount, topDataPtr, topNPerSheet, bottomDataPtr, bottomNPerSheet,
         coord);
    break;
  case BILINEAR:
    bilinear_propagation_kernel<Dtype> << <CAFFE_GET_BLOCKS(nCount),
                                           CAFFE_CUDA_NUM_THREADS>>>
        (nCount, topDataPtr, topNPerSheet, bottomDataPtr, bottomNPerSheet,
         coord, bottom->width());
    break;
  default:
    LOG(ERROR) << "Unknown interpolation mode " << interp;
  }
  CUDA_POST_KERNEL_CHECK;
}

// Explicit instantiation
template void PropagateErrorNN_gpu<float>(const Blob<float> *top,
                                          const float *coord,
                                          Blob<float> *bottom,
                                          const Interp &interp);
// template void PropagateErrorNN_gpu<double>(const Blob<double> *top,
//                                            const float *coord,
//                                            Blob<double> *bottom,
//                                            const Interp &interp);

template <typename Dtype>
__global__ void MaxTransSetSwitch_kernel(const Dtype *A, Dtype *B, int count,
                                         const float *coord, int sheet_count,
                                         float *switchPtr, int tIndex) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < count) {
    // only update max switch if this coord_data is valid
    if (coord[index % sheet_count] >= 0 && A[index] > B[index]) {
      B[index] = A[index];
      switchPtr[index] = tIndex;
    }
  }
}

template <typename Dtype>
void MaxTransSetSwitch_gpu(const Dtype *A, Dtype *B, int count,
                           const float *coord, int sheet_count, float *switchD,
                           int tIndex) {
  MaxTransSetSwitch_kernel<Dtype> << <CAFFE_GET_BLOCKS(count),
                                      CAFFE_CUDA_NUM_THREADS>>>
      (A, B, count, coord, sheet_count, switchD, tIndex);
}

// Explicit instantiation
template void MaxTransSetSwitch_gpu<float>(const float *A, float *B, int count,
                                           const float *coord, int sheet_count,
                                           float *switchD, int tIndex);
// template void MaxTransSetSwitch_gpu<double>(const double *A, double *B,
//                                             const int count, float *switchD,
//                                             const int tIndex);

template <typename Dtype>
__global__ void
ErrorPropagateDownpoolNN_kernel(const Dtype *topDiff, const int topCount,
                                const int topSheetCount, const float *switchD,
                                float **coord, Dtype **bottomDiff,
                                const int *bottomSheetCount) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < topCount) {
    int tIndex = static_cast<int>(switchD[index]);
    if (tIndex == 0) { // identity transformation, no need to change index
      atomicAdd((&(bottomDiff[tIndex]))[index],
                static_cast<float>(topDiff[index]));
    } else {
      int sheetOffset = index % topSheetCount; // ind of w and h in top
      int backSheetOffset = static_cast<int>((coord[tIndex][sheetOffset]));
      if (backSheetOffset >= 0) {
        int numSheet = index / topSheetCount; // channel*num
        int backBlobOffset =
            (bottomSheetCount[tIndex]) * numSheet + backSheetOffset;
        // AJ: same issue with lack of double atomicAdd
        atomicAdd((&(bottomDiff[tIndex]))[backBlobOffset],
                  static_cast<float>(topDiff[index]));
      }
    }
  }
}

template <typename Dtype>
void ErrorPropagateDownpoolNN_gpu(const Dtype *topDiff, const int topCount,
                                  const int topSheetCount, const float *switchD,
                                  float **coord, Dtype **bottomDiff,
                                  const int *bottomSheetCount) {
  ErrorPropagateDownpoolNN_kernel<Dtype> << <CAFFE_GET_BLOCKS(topCount),
                                             CAFFE_CUDA_NUM_THREADS>>>
      (topDiff, topCount, topSheetCount, switchD, coord, bottomDiff,
       bottomSheetCount);
}

// Explicit instantiation
template void ErrorPropagateDownpoolNN_gpu<float>(
    const float *topDiff, int topCount, int topSheetCount, const float *switchD,
    float **coord, float **bottomDiff, const int *bottomSheetCount);
// template void ErrorPropagateDownpoolNN_gpu<double>(
//     const float *topDiff, int topCount, int topSheetCount, const float
// *switchD,
//     float **coord, double **bottomDiff, const int *bottomSheetCount);

// For downpool layer, uses max switch.
template <typename Dtype>
__global__ void
nn_error_propagation_kernel_single(const Dtype *top, const int t_id,
                                   const int topCount, const int topSheetCount,
                                   const float *switchD, const float *coord,
                                   Dtype *bottom, const int bottomSheetCount) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < topCount) {
    int tIndex = static_cast<int>(switchD[index]);
    if (tIndex == t_id) { // Only do this if tIndex is at t_id
      if (tIndex == 0) { // identity transformation, no need to change anything,
                         // atomic is not necessary in this version only
        bottom[index] += top[index];
      } else {
        int sheetOffset = index % topSheetCount; // ind of w and h in top
        int backSheetOffset = static_cast<int>((coord[sheetOffset]));
        if (backSheetOffset >= 0) {
          int numSheet = index / topSheetCount; // channel*num
          int backBlobOffset = (bottomSheetCount) * numSheet + backSheetOffset;
          // AJ: same issue with lack of double atomicAdd
          atomicAdd(&bottom[backBlobOffset], static_cast<float>(top[index]));
        }
      }
    } // end of tIndex == t_id
  }
} // end of nn_error_propagation_kernel_single

// For downpool layer, uses max switch to propagate error.
template <typename Dtype>
__global__ void bilinear_error_propagation_kernel_single(
    const Dtype *top, const int t_id, const int topCount,
    const int topSheetCount, const float *switchD, const float *coord,
    Dtype *bottom, const int bottomSheetCount, int W) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < topCount) {
    int tIndex = static_cast<int>(switchD[index]);
    if (tIndex == t_id) { // Only do this if tIndex is at t_id
      if (tIndex == 0) {
        bottom[index] += top[index];
      } else {
        int sheetOffset = index % topSheetCount; // ind of w and h in top
        int backSheetOffset = static_cast<int>((coord[sheetOffset]));
        if (backSheetOffset >= 0) {
          int c0 = backSheetOffset % W;
          int ind_p11 = static_cast<int>(coord[sheetOffset + topSheetCount]);
          int c1 = ind_p11 % W;

          int ind_p01 = backSheetOffset - c0 + c1; // r0*W + c1
          int ind_p10 = ind_p11 - c1 + c0;         // r1*W  + c0

          float dc = coord[sheetOffset + 2 * topSheetCount];
          float dr = coord[sheetOffset + 3 * topSheetCount];

          float w00 = (1 - dc) * (1 - dr);
          float w01 = (1 - dr) * dc;
          float w10 = (1 - dc) * dr;
          float w11 = dr * dc;

          int numSheet = index / topSheetCount; // channel*num
          float top_error = static_cast<float>(top[index]);

          int commonOffset = numSheet * bottomSheetCount;

          atomicAdd(&bottom[commonOffset + backSheetOffset],
                    w00 * top_error);                                  // p00
          atomicAdd(&bottom[commonOffset + ind_p01], w01 * top_error); // p01
          atomicAdd(&bottom[commonOffset + ind_p10], w10 * top_error); // p10
          atomicAdd(&bottom[commonOffset + ind_p11], w11 * top_error); // p11
        }
      }
    } // end of tIndex == t_id
  }
} // end of bilinear_error_propagation_kernel_single

// AJ ErrorPropagateDownpoolNN_gpu that runs on each transformation separately
// t_id is the one that is being operated now
template <typename Dtype>
void ErrorPropagateDownpoolNN_gpu_single(
    const Dtype *top, const int &t_id, const int &topCount,
    const int &topSheetCount, const float *switchD, const float *coord,
    Dtype *bottom, const int &bottomSheetCount, const int &width,
    const Interp &interp) {
  switch (interp) {
  case NN:
    nn_error_propagation_kernel_single
            << <CAFFE_GET_BLOCKS(topCount), CAFFE_CUDA_NUM_THREADS>>>
        (top, t_id, topCount, topSheetCount, switchD, coord, bottom,
         bottomSheetCount);
    break;
  case BILINEAR:
    bilinear_error_propagation_kernel_single
            << <CAFFE_GET_BLOCKS(topCount), CAFFE_CUDA_NUM_THREADS>>>
        (top, t_id, topCount, topSheetCount, switchD, coord, bottom,
         bottomSheetCount, width);
    break;
  default:
    LOG(ERROR) << "Unknown interpolation mode " << interp;
  }
  CUDA_POST_KERNEL_CHECK;
}

// Explicit instantiation
template void ErrorPropagateDownpoolNN_gpu_single<float>(
    const float *top, const int &t_id, const int &topCount,
    const int &topSheetCount, const float *switchD, const float *coord,
    float *bottom, const int &bottomSheetCount, const int &width,
    const Interp &interp);

// Counts the usage of transformation:
//  - switch_data is a raw pointer to device memory.
//  - counter_data is a pointer with HOST memory.
void CountSwitches(float *switch_data, int n, int num_t, int *counter) {
  // wrap the raw pointer
  const thrust::device_ptr<float> d_ptr(switch_data);
  thrust::device_vector<float> vec(d_ptr, d_ptr + n);
  for (int t = 0; t < num_t; ++t) {
    counter[t] = thrust::count(vec.begin(), vec.end(), t);
  }
}

} // end namespace
